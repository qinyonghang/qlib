#include <hip/hip_runtime.h>
#include <iostream>

// constexpr struct config {
//     int gird_size;
//     int block_size;

//     constexpr config() {
//         cudaOccupancyMaxPotentialBlockSize
//     }
// } g_config;

template <class T>
__global__ void add(T *dst, T *src1, T src2) {
    auto i = threadIdx.x + blockIdx.x * blockDim.x;
    dst[i] = src1[i] + src2;
}

template <class T>
__global__ void add(T *dst, T *src1, T *src2) {
    auto i = threadIdx.x + blockIdx.x * blockDim.x;
    dst[i] = src1[i] + src2[i];
}

template <class T>
__global__ void sub(T *dst, T *src1, T src2) {
    auto i = threadIdx.x + blockIdx.x * blockDim.x;
    dst[i] = src1[i] - src2;
}

template <class T>
__global__ void sub(T *dst, T *src1, T *src2) {
    auto i = threadIdx.x + blockIdx.x * blockDim.x;
    dst[i] = src1[i] - src2[i];
}

template <class T>
__global__ void mul(T *dst, T *src1, T src2) {
    auto i = threadIdx.x + blockIdx.x * blockDim.x;
    dst[i] = src1[i] * src2;
}

template <class T>
__global__ void div(T *dst, T *src1, T src2) {
    auto i = threadIdx.x + blockIdx.x * blockDim.x;
    dst[i] = src1[i] / src2;
}

template <class T>
__global__ void mod(T *dst, T *src1, T *src2) {
    auto i = threadIdx.x + blockIdx.x * blockDim.x;
    dst[i] = src1[i] % src2[i];
}

__global__ void vectorAddKernel(int* a, int* b, int* result) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    result[i] = a[i] + b[i];
}

void vectorAdd(int* h_a, int* h_b, int* h_result, int n) {
    const auto aligned_n = ((n + 255) / 256) * 256;
    int *d_a, *d_b, *d_result;

    hipMalloc(&d_a, aligned_n * sizeof(int));
    hipMalloc(&d_b, aligned_n * sizeof(int));
    hipMalloc(&d_result, aligned_n * sizeof(int));
    // cudaMallocManaged(&d_a, aligned_n * sizeof(int));
    // cudaMallocManaged(&d_b, aligned_n * sizeof(int));
    // cudaMallocManaged(&d_result, aligned_n * sizeof(int));

    // memcpy(d_a, h_a, n * sizeof(int));
    // memcpy(d_b, h_b, n * sizeof(int));
    hipMemcpy(d_a, h_a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    vectorAddKernel<<<aligned_n / 256, 256>>>(d_a, d_b, d_result);
    hipEventRecord(stop);

    hipDeviceSynchronize();

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << "GPU kernel time: " << milliseconds << " ms" << std::endl;

    // memcpy(h_result, d_result, n * sizeof(int));
    hipMemcpy(h_result, d_result, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
}
